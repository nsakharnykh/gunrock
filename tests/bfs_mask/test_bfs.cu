// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

// Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <algorithm>
#include <iostream>
#include <fstream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>
#include <gunrock/util/track_utils.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;

void ref_bfs_mask(const int src_node, const int dst_node, const int num_nodes, const int num_edges, const int *row_offsets, const int *col_indices, const int *col_mask, int *parents)
{
  int *q = (int*)malloc(num_nodes * sizeof(int));
  q[0] = src_node;
  parents[src_node] = src_node;
  int idx = -1;
  int size = 1;
  int found = 0;
  while (idx+1 < size && !found) {
    idx++;
    int u = q[idx];
    for (int i = row_offsets[u]; i < row_offsets[u+1]; i++) {
      int v = col_indices[i];
      if (parents[v] == -1 && col_mask[i]) {
        parents[v] = u;
        if (v == dst_node) {
          found = 1;
          break;
        }
        else {
          q[size] = v;
          size++;
        }
      }
    }
  }
}

hipError_t bfs_mask(int src_node, int dst_node, int num_nodes, int num_edges, int *row_offsets, int *col_indices, int *col_mask, int *parents)
{
  // TODO: use Gunrock's customized BFS here
  //ref_bfs_mask(src_node, dst_node, num_nodes, num_edges, row_offsets, col_indices, col_mask, parents);
  typedef int VertexId;
  typedef int SizeT;
  typedef int Value;
  typedef BFSProblem <VertexId,SizeT,Value,
                      true, // MARK_PREDECESSORS
                      true> // IDEMPOTENCE
                      Problem;
  typedef BFSEnactor <Problem> Enactor;

  hipError_t retval = hipSuccess;

  Info<VertexId, SizeT, Value> *info = new Info<VertexId, SizeT, Value>;

  
  info->InitBase2("BFS");
  ContextPtr *context = (ContextPtr*)info->context;
  hipStream_t *streams = (hipStream_t*)info->streams;

  int *gpu_idx = new int[1];
  gpu_idx[0] = 0;

  Problem *problem = new Problem(false, false); //no direction optimized, no undirected
  if (retval = util::GRError(problem->Init(
    false, //stream_from_host (depricated)
    row_offsets,
    col_indices,
    col_mask,
    parents,
    num_nodes,
    num_edges,
    1,
    NULL,
    "random",
    streams),
    "BFS Problem Init failed", __FILE__, __LINE__)) return retval;

  Enactor *enactor = new Enactor(1, gpu_idx);

  if (retval = util::GRError(enactor->Init(context, problem),
  "BFS Enactor Init failed.", __FILE__, __LINE__)) return retval;

  if (retval = util::GRError(problem->Reset(
  src_node, enactor->GetFrontierType()),
  "BFS Problem Reset failed", __FILE__, __LINE__))
  return retval;

  if (retval = util::GRError(enactor->Reset(),
  "BFS Enactor Reset failed", __FILE__, __LINE__))
  return retval;

  if (retval = util::GRError(enactor->Enact(src_node),
  "BFS Enact failed", __FILE__, __LINE__)) return retval;

  return retval;
}

int main(int argc, char** argv)
{
  // initialize graph here
  int num_nodes = 7, num_edges = 14, src_node = 0, dst_node = 6;
  int row_offsets[8]  = {0, 2, 5, 8, 10, 13, 14, 14};
  int col_indices[14] = {1, 2, 0, 2, 4, 3, 4, 5, 5, 6, 2, 5, 6, 6};
  int col_mask[14]    = {1, 1, 1, 0, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1};

  // setup memory in gpu
  int *d_row_offsets;
  hipMallocManaged(&d_row_offsets, (num_nodes + 1) * sizeof(int));
  memcpy(d_row_offsets, row_offsets, (num_nodes + 1) * sizeof(int));
  int *d_col_indices;
  hipMallocManaged(&d_col_indices, num_edges * sizeof(int));
  memcpy(d_col_indices, col_indices, num_edges * sizeof(int));
  int *d_col_mask;
  hipMallocManaged(&d_col_mask, num_edges * sizeof(int));
  memcpy(d_col_mask, col_mask, num_edges * sizeof(int));
  int *d_parents;
  hipMallocManaged(&d_parents, num_nodes * sizeof(int));
  for (int node = 0; node < num_nodes; node++) 
    d_parents[node] = -1;

  // run bfs (with mask)
  bfs_mask(src_node, dst_node, num_nodes, num_edges, d_row_offsets, d_col_indices, d_col_mask, d_parents);

  // print out results
  for (int node = 0; node < num_nodes; node++)
    printf("Node_ID [%d] : Parent [%d]\n", node, d_parents[node]);

  // free memory
  hipFree(d_row_offsets);
  hipFree(d_col_indices);
  hipFree(d_col_mask);
  hipFree(d_parents);

  return 0;
}
// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
